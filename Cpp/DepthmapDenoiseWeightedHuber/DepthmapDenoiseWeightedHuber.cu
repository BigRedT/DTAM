#include "hip/hip_runtime.h"
#include <opencv2/gpu/device/common.hpp>//for cudaSafeCall
#include <opencv2/core/core.hpp>//for CV_Assert
#include "DepthmapDenoiseWeightedHuber.cuh"

namespace cv { namespace gpu { namespace device {
    namespace dtam_denoise{


static unsigned int arows;//TODO:make sure this is still reentrant

void loadConstants(uint h_rows, uint, uint , uint ,
        float* , float* , float* , float* , float* ,
        float*) {

        arows=h_rows;
}

hipStream_t localStream=0;

const int BLOCKX2D=32;
const int BLOCKY2D=32;
#define GENERATE_CUDA_FUNC2D(funcName,arglist,notypes)                                     \
static __global__ void funcName arglist;                                                        \
void funcName##Caller arglist{                                                           \
   dim3 dimBlock(BLOCKX2D,BLOCKY2D);                                                                  \
   dim3 dimGrid((acols  + dimBlock.x - 1) / dimBlock.x,                                  \
                (arows + dimBlock.y - 1) / dimBlock.y);                                  \
   funcName<<<dimGrid, dimBlock,0,localStream>>>notypes;                                  \
   cudaSafeCall( hipGetLastError() );\
};static __global__ void funcName arglist


#define GENERATE_CUDA_FUNC2DROWS(funcName,arglist,notypes)                                     \
static __global__ void funcName arglist;                                                        \
void funcName##Caller arglist{                                                           \
   dim3 dimBlock(BLOCKX2D,BLOCKY2D);                                                                  \
   dim3 dimGrid(1,                                  \
                (arows + dimBlock.y - 1) / dimBlock.y);                                  \
   funcName<<<dimGrid, dimBlock,0,localStream>>>notypes;                                  \
   cudaSafeCall( hipGetLastError() );\
};static __global__ void funcName arglist


static __global__ void computeG1  (float* pp, float* g1p, float* gxp, float* gyp, int cols);
static __global__ void computeG2  (float* pp, float* g1p, float* gxp, float* gyp, int cols);
void computeGCaller  (float* pp, float* g1p, float* gxp, float* gyp, int cols){
//   dim3 dimBlock(BLOCKX2D,BLOCKY2D);
   dim3 dimBlock(BLOCKX2D,4);
   dim3 dimGrid(1,
                (arows + dimBlock.y - 1) / dimBlock.y);

   computeG1<<<dimGrid, dimBlock,0,localStream>>>(pp, g1p, gxp, gyp, cols);
   hipDeviceSynchronize();
   computeG2<<<dimGrid, dimBlock,0,localStream>>>(pp, g1p, gxp, gyp, cols);
   hipDeviceSynchronize();
   
   cudaSafeCall( hipGetLastError() );
};

GENERATE_CUDA_FUNC2DROWS(computeG1,
                     (float* pp, float* g1p, float* gxp, float* gyp, int cols),
                     (pp, g1p, gxp, gyp, cols)) {
    #if __CUDA_ARCH__>=300
//TODO: make compatible with cuda 2.0 and lower (remove shuffles). Probably through texture fetch

//Original pseudocode for this function:
    // //subscripts u,d,l,r mean up,down,left,right
    // void computeG(){
    //     // g0 is the strongest nearby gradient (excluding point defects)
    //     g0x=fabsf(pr-pl);//|dx|
    //     g0y=fabsf(pd-pu);//|dy|
    //     g0=max(g0x,g0y);
    //     // g1 is the scaled g0 through the g function exp(-alpha*x^beta)
    //     g1=sqrt(g0); //beta=0.5
    //     alpha=3.5;
    //     g1=exp(-alpha*g1);
    //     //hard to explain this without a picture, but breaks are where both neighboring pixels are near a change
    //     gx=max(g1r,g1);
    //     gy=max(g1d,g1);
    //     gu=gyu;  //upper spring is the lower spring of the pixel above
    //     gd=gy;   //lower spring
    //     gr=gx;   //right spring
    //     gl=gxl;  //left spring is the right spring of the pixel to the left
    // }
    const float alpha=3.5f;
    int x = threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int upoff=-(y!=0)*cols;
    int dnoff=(y<gridDim.y*blockDim.y-1)*cols;
    //itr0
    int pt=x+y*cols;
    float ph,pn,pu,pd,pl,pr;
    float g0x,g0y,g0,g1,gt,gsav;
    float tmp;
    ph=pp[pt];
    pn=pp[pt+blockDim.x];

    pr=__shfl_down(ph,2);
    tmp=__shfl_up(pn,30);
    if(threadIdx.x>=30){
        pr=tmp;
    }
    pl=ph;
    pu=pp[pt+upoff];
    pd=pp[pt+dnoff];


    // g0 is the strongest nearby gradient (excluding point defects)
        gt=fabsf(pr-pl);
        g0x=__shfl_up(gt,1);//?xxxxxx no prior val
        gsav=__shfl_down(gt,31);//x000000 for next time
        g0x=threadIdx.x>0?g0x:0.0f;//0xxxxxx
        g0y=fabsf(pd-pu);

        g0=fmaxf(g0x,g0y);
    // g1 is the scaled g0 through the g function
        g1=sqrt(g0);
        g1=exp(-alpha*g1);
    //save
        g1p[pt]=g1;

    x+=32;
    //itr 1:n-2
    for(;x<cols-32;x+=32){
        pt=x+y*cols;
        ph=pn;
        pn=pp[pt+blockDim.x];
        pr=__shfl_down(ph,2);
        tmp=__shfl_up(pn,30);
        pr=threadIdx.x>=30?tmp:pr;

        pl=ph;
        pu=pp[pt+upoff];
        pd=pp[pt+dnoff];

        // g0 is the strongest nearby gradient (excluding point defects)
            gt=fabsf(pr-pl);
            g0x=__shfl_up(gt,1);//?xxxxxx
            g0x=threadIdx.x>0?g0x:gsav;//xxxxxxx
            gsav=__shfl_down(gt,31);//x000000 for next time
            g0y=fabsf(pd-pu);

            g0=fmaxf(g0x,g0y);

        // g1 is the scaled g0 through the g function
            g1=sqrt(g0);
            g1=exp(-alpha*g1);
        //save
            g1p[pt]=g1;
    }

    //itr n-1
    pt=x+y*cols;
    ph=pn;
    pr=__shfl_down(ph,2);
    pl=ph;
    pu=pp[pt+upoff];
    pd=pp[pt+dnoff];

    // g0 is the strongest nearby gradient (excluding point defects)
        gt=fabsf(pr-pl);
        g0x=__shfl_up(gt,1);//?xxxxxx
        g0x=threadIdx.x>0?g0x:gsav;//xxxxxxx
        g0y=fabsf(pd-pu);

        g0=fmaxf(g0x,g0y);
    // g1 is the scaled g0 through the g function
        g1=sqrt(g0);
        g1=exp(-alpha*g1);
    //save
        g1p[pt]=g1;
#endif
}
GENERATE_CUDA_FUNC2DROWS(computeG2,
                     (float* pp, float* g1p, float* gxp, float* gyp, int cols),
                     (pp, g1p, gxp, gyp, cols)) {
    #if __CUDA_ARCH__>=300
    int x = threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int dnoff=(y<gridDim.y*blockDim.y-1)*cols;
    //itr0
    int pt=x+y*cols;
    float g1h,g1n,g1u,g1d,g1r,g1l,gx,gy;
    float tmp;
//part2, find gx,gy
    x = threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;
    //itr0
    pt=x+y*cols;

    g1h=g1p[pt];
    g1n=g1p[pt+blockDim.x];
    g1r=__shfl_down(g1h,1);
    tmp=__shfl_up(g1n,31);
    if(threadIdx.x>=31){
        g1r=tmp;
    }
    g1l=g1h;
    g1u=g1h;
    g1d=g1p[pt+dnoff];

    gx=fmaxf(g1l,g1r);
    gy=fmaxf(g1u,g1d);

    //save
        gxp[pt]=gx;
        gyp[pt]=gy;
    x+=32;
    //itr 1:n-2
    for(;x<cols-32;x+=32){
        pt=x+y*cols;
        g1h=g1n;
        g1n=g1p[pt+blockDim.x];
        g1r=__shfl_down(g1h,1);
        tmp=__shfl_up(g1n,31);
        g1r=threadIdx.x>=31?tmp:g1r;

        g1l=g1h;
        g1u=g1h;
        g1d=g1p[pt+dnoff];

        gx=fmaxf(g1l,g1r);
        gy=fmaxf(g1u,g1d);
        //save
            gxp[pt]=gx;
            gyp[pt]=gy;
    }

    //itr n-1
    pt=x+y*cols;
    g1h=g1n;
    g1r=__shfl_down(g1h,1);
    g1l=g1h;
    g1u=g1h;
    g1d=g1p[pt+dnoff];

    gx=fmaxf(g1l,g1r);
    gy=fmaxf(g1u,g1d);


    //save
        gxp[pt]=gx;
        gyp[pt]=gy;
#endif
}


//This version is faster, but makes synchronization errors at the lines between parts 1 and 2.
//Could be fixed by a second pass for part 2 over the stitch lines, but I don't have time to figure that out
//right now.
GENERATE_CUDA_FUNC2DROWS(computeGunsafe,
                     (float* pp, float* g1p, float* gxp, float* gyp, int cols),
                     (pp, g1p, gxp, gyp, cols)) {
    #if __CUDA_ARCH__>=300
//TODO: make compatible with cuda 2.0 and lower (remove shuffles). Probably through texture fetch
//TODO: rerun kernel on lines with y%32==31 or y%32==0 to fix stitch lines

//Original pseudocode for this function:
    // //subscripts u,d,l,r mean up,down,left,right
    // void computeG(){
    //     // g0 is the strongest nearby gradient (excluding point defects)
    //     g0x=fabsf(pr-pl);//|dx|
    //     g0y=fabsf(pd-pu);//|dy|
    //     g0=max(g0x,g0y);
    //     // g1 is the scaled g0 through the g function exp(-alpha*x^beta)
    //     g1=sqrt(g0); //beta=0.5
    //     alpha=3.5;
    //     g1=exp(-alpha*g1);
    //     //hard to explain this without a picture, but breaks are where both neighboring pixels are near a change
    //     gx=max(g1r,g1);
    //     gy=max(g1d,g1);
    //     gu=gyu;  //upper spring is the lower spring of the pixel above
    //     gd=gy;   //lower spring
    //     gr=gx;   //right spring
    //     gl=gxl;  //left spring is the right spring of the pixel to the left
    // }
    const float alpha=3.5f;
    int x = threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int upoff=-(y!=0)*cols;
    int dnoff=(y<gridDim.y*blockDim.y-1)*cols;
    //itr0
    int pt=x+y*cols;
    float ph,pn,pu,pd,pl,pr;
    float g0x,g0y,g0,g1,g1h,g1n,g1u,g1d,g1r,g1l,gx,gy,gt,gsav;
    float tmp;
    ph=pp[pt];
    pn=pp[pt+blockDim.x];

    pr=__shfl_down(ph,2);
    tmp=__shfl_up(pn,30);
    if(threadIdx.x>=30){
        pr=tmp;
    }
    pl=ph;
    pu=pp[pt+upoff];
    pd=pp[pt+dnoff];


    // g0 is the strongest nearby gradient (excluding point defects)
        gt=fabsf(pr-pl);
        g0x=__shfl_up(gt,1);//?xxxxxx no prior val
        gsav=__shfl_down(gt,31);//x000000 for next time
        g0x=threadIdx.x>0?g0x:0.0f;//0xxxxxx
        g0y=fabsf(pd-pu);

        g0=fmaxf(g0x,g0y);
    // g1 is the scaled g0 through the g function
        g1=sqrt(g0);
        g1=exp(-alpha*g1);
    //save
        g1p[pt]=g1;

    x+=32;
    //itr 1:n-2
    for(;x<cols-32;x+=32){
        pt=x+y*cols;
        ph=pn;
        pn=pp[pt+blockDim.x];
        pr=__shfl_down(ph,2);
        tmp=__shfl_up(pn,30);
        pr=threadIdx.x>=30?tmp:pr;

        pl=ph;
        pu=pp[pt+upoff];
        pd=pp[pt+dnoff];

        // g0 is the strongest nearby gradient (excluding point defects)
            gt=fabsf(pr-pl);
            g0x=__shfl_up(gt,1);//?xxxxxx
            g0x=threadIdx.x>0?g0x:gsav;//xxxxxxx
            gsav=__shfl_down(gt,31);//x000000 for next time
            g0y=fabsf(pd-pu);

            g0=fmaxf(g0x,g0y);

        // g1 is the scaled g0 through the g function
            g1=sqrt(g0);
            g1=exp(-alpha*g1);
        //save
            g1p[pt]=g1;
    }

    //itr n-1
    pt=x+y*cols;
    ph=pn;
    pr=__shfl_down(ph,2);
    pl=ph;
    pu=pp[pt+upoff];
    pd=pp[pt+dnoff];

    // g0 is the strongest nearby gradient (excluding point defects)
        gt=fabsf(pr-pl);
        g0x=__shfl_up(gt,1);//?xxxxxx
        g0x=threadIdx.x>0?g0x:gsav;//xxxxxxx
        g0y=fabsf(pd-pu);

        g0=fmaxf(g0x,g0y);
    // g1 is the scaled g0 through the g function
        g1=sqrt(g0);
        g1=exp(-alpha*g1);
    //save
        g1p[pt]=g1;

//part2, find gx,gy
    x = threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;
    //itr0
    pt=x+y*cols;

    g1h=g1p[pt];
    g1n=g1p[pt+blockDim.x];
    g1r=__shfl_down(g1h,1);
    tmp=__shfl_up(g1n,31);
    if(threadIdx.x>=31){
        g1r=tmp;
    }
    g1l=g1h;
    g1u=g1h;
    g1d=g1p[pt+dnoff];

    gx=fmaxf(g1l,g1r);
    gy=fmaxf(g1u,g1d);

    //save
        gxp[pt]=gx;
        gyp[pt]=gy;
    x+=32;
    //itr 1:n-2
    for(;x<cols-32;x+=32){
        pt=x+y*cols;
        g1h=g1n;
        g1n=g1p[pt+blockDim.x];
        g1r=__shfl_down(g1h,1);
        tmp=__shfl_up(g1n,31);
        g1r=threadIdx.x>=31?tmp:g1r;

        g1l=g1h;
        g1u=g1h;
        g1d=g1p[pt+dnoff];

        gx=fmaxf(g1l,g1r);
        gy=fmaxf(g1u,g1d);
        //save
            gxp[pt]=gx;
            gyp[pt]=gy;
    }

    //itr n-1
    pt=x+y*cols;
    g1h=g1n;
    g1r=__shfl_down(g1h,1);
    g1l=g1h;
    g1u=g1h;
    g1d=g1p[pt+dnoff];

    gx=fmaxf(g1l,g1r);
    gy=fmaxf(g1u,g1d);


    //save
        gxp[pt]=gx;
        gyp[pt]=gy;
#endif

}
__device__ inline float saturate(float x){
    //return x;
    return x/fmaxf(1.0f,fabsf(x));
}
// static __global__ void updateQD  (float* gqxpt, float* gqypt, float *dpt, float * apt,
//                float *gxpt, float *gypt, float sigma_q, float sigma_d, float epsilon,
//                float theta);//DANGER, no interblock synchronization = weird instability
static __global__ void updateQ  (float* gqxpt, float* gqypt, float *dpt, float * apt,
                float *gxpt, float *gypt, int cols, float sigma_q, float sigma_d, float epsilon,
                float theta);
static __global__ void updateD  (float* gqxpt, float* gqypt, float *dpt, float * apt,
                float *gxpt, float *gypt, int cols, float sigma_q, float sigma_d, float epsilon,
                float theta);

void updateQDCaller(float* gqxpt, float* gqypt, float *dpt, float * apt,
        float *gxpt, float *gypt, int cols, float sigma_q, float sigma_d, float epsilon,
        float theta) {

    dim3 dimBlock(BLOCKX2D, BLOCKY2D);
    dim3 dimGrid(1, (arows + dimBlock.y - 1) / dimBlock.y);
    CV_Assert(dimGrid.y>0);
    cudaSafeCall( hipGetLastError() );
    updateQ<<<dimGrid, dimBlock,0,localStream>>>( gqxpt, gqypt, dpt, apt,
            gxpt, gypt, cols, sigma_q, sigma_d, epsilon, theta);
    cudaSafeCall( hipGetLastError() );
    updateD<<<dimGrid, dimBlock,0,localStream>>>( gqxpt, gqypt, dpt, apt,
            gxpt, gypt, cols, sigma_q, sigma_d, epsilon, theta);
    cudaSafeCall( hipGetLastError() );
};

// static __global__ void updateQD  (float* gqxpt, float* gqypt, float *dpt, float * apt,
//                 float *gxpt, float *gypt, float sigma_q, float sigma_d, float epsilon,
//                 float theta) {
//     //TODO: make compatible with cuda 2.0 and lower (remove shuffles). Probably through texture fetch
// 
//     //Original pseudocode for this function:
// //void updateQD(){
// //    //shifts are shuffles!
// //    for (all x in blocks of warpsize;;){
// //        //qx update
// //        float dh,dn,qxh,gx,gqx,qyh,gy,gqy;
// //        //load(dh,dn,gxh,gqx);//load here, next(the block to the right), local constant, old x force(with cached multiply)
// //        dr=dh<<1;
// //        tmp=dn>>31;
// //        if (rt)
// //            dr=tmp;
// //        qxh=gqx/gxh;
// //        qxh = (qxh+sigma_q*gxh*(dr-dh))/(1+sigma_q*epsilon);//basic spring force equation f=k(x-x0)
// //        gqx = saturate(gxh*qxh);//spring saturates (with cached multiply), saturation force proportional to prob. of not an edge.
// //        gqxpt[pt]=gqx;
// //
// //        //qy update
// //        s[bpt]=dn;
// //        if(!btm){
// //            dd=s[bpt+bdnoff];
// //        }else{
// //            dd=dpt[pt+dnoff];
// //        }
// //        qyh=gqy/gy;
// //        qyh=(qyh+sigma_q*gyh*(dd-dh))/(1+sigma_q*epsilon);
// //        gqy=saturate(gyh*qyh);
// //        gqypt[pt]=gqy;
// //
// //        //dx update
// //        gqr=gqx;
// //        gql=gqx>>1;
// //        if (lf)
// //            gql=gqsave;
// //        gqsave=gqx<<31;//save for next iter
// //        dacc = gqr - gql;//dx part
// //
// //        //dy update and d store
// //        gqd=gqy;
// //        s[bpt]=gqy;
// //        if(!top)
// //            gqu=s[bpt+bupoff];
// //        else
// //            gqu=gqxpt[pt + upoff];
// //        dacc += gqd-gqu; //dy part
// //        d = (d + sigma_d*(dacc+1/theta*ah))/(1+sigma_d/theta);
// //        dpt[pt]=d;
// //    }
// //}
//     __shared__ float s[32*BLOCKY2D];
//     int x = threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     bool rt=x==31;
//     bool lf=x==0;
//     bool top=y==0;
//     bool btm=y==rows-1;
//     bool btop=threadIdx.y==0;
//     bool bbtm=threadIdx.y==blockDim.y-1;
//     int pt, bpt,bdnoff ,dnoff, bupoff, upoff;
// 
// 
//     float tmp,gqsave;
//     gqsave=0;
//     bpt = threadIdx.x+threadIdx.y*blockDim.x;
//     bdnoff=blockDim.x;
//     dnoff=(!btm)*cols;
//     bupoff=-blockDim.x;
//     upoff=-(!top)*cols;
// 
//     pt=x+y*cols;
// 
//     float dh,dn;
//     dn=dpt[pt];
// 
//     for(;x<cols;x+=32){
//         float qx,gx,gqx,qy,gy,gqy;
//         pt=x+y*cols;
// 
// 
//         //qx update
//         {
//             float dr;
//             //load(dh,dn,gxh,gqx);//load here, next(the block to the right), local constant, old x force(with cached multiply)
// 
//             //load
//             {
//                 dh=dn;
//                 if(x<cols-32){
//                     dn=dpt[pt+32];
// 
//                 }
//                 gqx=gqxpt[pt];
//                 gx=gxpt[pt];
// //                gx=1.0f;
// 
//             }
// 
//             dr=__shfl_down(dh,1);
//             tmp=__shfl_up(dn,31);
//             if (rt && x<cols-32)
//                 dr=tmp;
//             qx = gqx/gx;
//             qx = (qx+sigma_q*gx*(dr-dh))/(1+sigma_q*epsilon);//basic spring force equation f=k(x-x0)
//             gqx = saturate(gx*qx);//spring saturates (with cached multiply), saturation force proportional to prob. of not an edge.
//             //gqxpt[pt]=gqx;
//         }
//         dpt[pt] = dh;
//         //qy update
//         {
//             float dd;
//             //load
//                     {
//                         gqy=gqypt[pt];
//                         gy=gypt[pt];
// //                        gy=1.0f;
//                     }
//             s[bpt]=dh;
//             __syncthreads();
//             if(!bbtm){
//                 dd=s[bpt+bdnoff];
//             }else{
//                 dd=dpt[pt+dnoff];
//             }
//             qy = gqy/gy;
//             qy = (qy+sigma_q*gy*(dd-dh))/(1+sigma_q*epsilon);
//             gqy = saturate(gy*qy);
//             //gqypt[pt]=gqy;
//         }
//         float dacc;
//         //dx update
//         {
//             float gqr,gql;
//             gqr=gqx;
//             gql=__shfl_up(gqx,1);
//             if (lf)
//                 gql=gqsave;
//             gqsave=__shfl_down(gqx,31);//save for next iter
//             dacc = gqr - gql;//dx part
//         }
//         float d=dh;
//         //dy update and d store
//         {
//             float a;
//             //load
//             {
//                 a=apt[pt];
//             }
//             float gqu,gqd;
// 
//             gqd=gqy;
//             s[bpt]=gqy;
//             __syncthreads();
//             if(!btop)
//                 gqu=s[bpt+bupoff];
//             else
//                 gqu=gqypt[pt + upoff];
//             if(y==0)
//                 gqu=0.0f;
//             dacc += gqd-gqu; //dy part
//             d = ( d + sigma_d*(dacc + a/theta) ) / (1 + sigma_d/theta);
//             //dpt[pt] = d;
//         }
//         __syncthreads();
//         gqxpt[pt]=gqx;
//         gqypt[pt]=gqy;
//         dpt[pt] = d;
//         __syncthreads();
//     }
// }


GENERATE_CUDA_FUNC2DROWS(updateQ,
                (float* gqxpt, float* gqypt, float *dpt, float * apt,
                float *gxpt, float *gypt, int cols, float sigma_q, float sigma_d, float epsilon,
                float theta),
                ( gqxpt, gqypt, dpt, apt,
                        gxpt, gypt, cols, sigma_q, sigma_d, epsilon, theta)) {
    //TODO: make compatible with cuda 2.0 and lower (remove shuffles). Probably through texture fetch

    //Original pseudocode for this function:
//void updateQD(){
//    //shifts are shuffles!
//    for (all x in blocks of warpsize;;){
//        //qx update
//        float dh,dn,qxh,gx,gqx,qyh,gy,gqy;
//        //load(dh,dn,gxh,gqx);//load here, next(the block to the right), local constant, old x force(with cached multiply)
//        dr=dh<<1;
//        tmp=dn>>31;
//        if (rt)
//            dr=tmp;
//        qxh=gqx/gxh;
//        qxh = (qxh+sigma_q*gxh*(dr-dh))/(1+sigma_q*epsilon);//basic spring force equation f=k(x-x0)
//        gqx = saturate(gxh*qxh);//spring saturates (with cached multiply), saturation force proportional to prob. of not an edge.
//        gqxpt[pt]=gqx;
//
//        //qy update
//        s[bpt]=dn;
//        if(!btm){
//            dd=s[bpt+bdnoff];
//        }else{
//            dd=dpt[pt+dnoff];
//        }
//        qyh=gqy/gy;
//        qyh=(qyh+sigma_q*gyh*(dd-dh))/(1+sigma_q*epsilon);
//        gqy=saturate(gyh*qyh);
//        gqypt[pt]=gqy;
//
//        //dx update
//        gqr=gqx;
//        gql=gqx>>1;
//        if (lf)
//            gql=gqsave;
//        gqsave=gqx<<31;//save for next iter
//        dacc = gqr - gql;//dx part
//
//        //dy update and d store
//        gqd=gqy;
//        s[bpt]=gqy;
//        if(!top)
//            gqu=s[bpt+bupoff];
//        else
//            gqu=gqxpt[pt + upoff];
//        dacc += gqd-gqu; //dy part
//        d = (d + sigma_d*(dacc+1/theta*ah))/(1+sigma_d/theta);
//        dpt[pt]=d;
//    }
//}
#if __CUDA_ARCH__>=300
    __shared__ float s[32*BLOCKY2D];
    int x = threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    bool rt=x==31;

    bool bbtm=threadIdx.y==blockDim.y-1;
    int pt, bpt,bdnoff ,dnoff;
    
    float tmp;
    bpt = threadIdx.x+threadIdx.y*blockDim.x;
    bdnoff=blockDim.x;
    dnoff=(y<gridDim.y*blockDim.y-1)*cols;

    pt=x+y*cols;

    float dh,dn;
    dn=dpt[pt];


    for(;x<cols;x+=32){
        float qx,gx,gqx,qy,gy,gqy;
        pt=x+y*cols;


        //qx update
        {
            float dr;
            //load(dh,dn,gxh,gqx);//load here, next(the block to the right), local constant, old x force(with cached multiply)

            //load
            {
                dh=dn;
                if(x<cols-32){
                    dn=dpt[pt+32];

                }
                gqx=gqxpt[pt];
                gx=gxpt[pt]+.005f;
//                gx=1.0f;
            }

            dr=__shfl_down(dh,1);
            tmp=__shfl_up(dn,31);
            if (rt && x<cols-32)
                dr=tmp;
            qx = gqx/gx;
            //qx+=(gx*(dr-dh)-epsilon*qx)*.5f;//simplified step
            qx = (qx+sigma_q*gx*(dr-dh))/(1+sigma_q*epsilon);//basic spring force equation f=k(x-x0)
            gqx = gx*saturate(qx);
//             gqx = saturate(gx*qx);//spring saturates (with cached multiply), saturation force proportional to prob. of not an edge.
            gqxpt[pt]=gqx;
        }

        //qy update
        {
            float dd;
            //load
                    {
                        gqy=gqypt[pt];
                        gy=gypt[pt]+.005f;
//                        gy=1.0f;
                    }
            s[bpt]=dh;
            __syncthreads();
            if(!bbtm)
                dd=s[bpt+bdnoff];
            else
                dd=dpt[pt+dnoff];
            __syncthreads();
            qy = gqy/gy;
            //qy+=(gy*(dd-dh)-epsilon*qy)*.5f;//simplified step
            qy = (qy+sigma_q*gy*(dd-dh))/(1+sigma_q*epsilon);
            gqy = gy*saturate(qy);
//             gqy = saturate(gy*qy);

            gqypt[pt]=gqy;
        }
        //__syncthreads();
    }
#endif
}

GENERATE_CUDA_FUNC2DROWS(updateD,
                (float* gqxpt, float* gqypt, float *dpt, float * apt,
                float *gxpt, float *gypt,int cols, float sigma_q, float sigma_d, float epsilon,
                float theta),
                ( gqxpt, gqypt, dpt, apt,
                        gxpt, gypt, cols, sigma_q, sigma_d, epsilon, theta)) {
    #if __CUDA_ARCH__>=300
    //TODO: make compatible with cuda 2.0 and lower (remove shuffles). Probably through texture fetch

    //Original pseudocode for this function:
//void updateQD(){
//    //shifts are shuffles!
//    for (all x in blocks of warpsize){
//        //qx update
//        float dh,dn,qxh,gx,gqx,qyh,gy,gqy;
//        //load(dh,dn,gxh,gqx);//load here, next(the block to the right), local constant, old x force(with cached multiply)
//        dr=dh<<1;
//        tmp=dn>>31;
//        if (rt)
//            dr=tmp;
//        qxh=gqx/gxh;
//        qxh = (qxh+sigma_q*gxh*(dr-dh))/(1+sigma_q*epsilon);//basic spring force equation f=k(x-x0)
//        gqx = saturate(gxh*qxh);//spring saturates (with cached multiply), saturation force proportional to prob. of not an edge.
//        gqxpt[pt]=gqx;
//
//        //qy update
//        s[bpt]=dn;
//        if(!btm){
//            dd=s[bpt+bdnoff];
//        }else{
//            dd=dpt[pt+dnoff];
//        }
//        qyh=gqy/gy;
//        qyh=(qyh+sigma_q*gyh*(dd-dh))/(1+sigma_q*epsilon);
//        gqy=saturate(gyh*qyh);
//        gqypt[pt]=gqy;
//
//        //dx update
//        gqr=gqx;
//        gql=gqx>>1;
//        if (lf)
//            gql=gqsave;
//        gqsave=gqx<<31;//save for next iter
//        dacc = gqr - gql;//dx part
//
//        //dy update and d store
//        gqd=gqy;
//        s[bpt]=gqy;
//        if(!top)
//            gqu=s[bpt+bupoff];
//        else
//            gqu=gqxpt[pt + upoff];
//        dacc += gqd-gqu; //dy part
//        d = (d + sigma_d*(dacc+1/theta*ah))/(1+sigma_d/theta);
//        dpt[pt]=d;
//    }
//}
    __shared__ float s[32*BLOCKY2D];
    int x = threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    bool lf=x==0;
    bool top=y==0;
    bool btop=threadIdx.y==0;
    int pt, bpt, bupoff, upoff;


    float gqsave=0;
    bpt = threadIdx.x+threadIdx.y*blockDim.x;

    bupoff=-blockDim.x;
    upoff=-(!top)*cols;

    pt=x+y*cols;

    for(;x<cols;x+=32){
        float gqx,gqy;
        pt=x+y*cols;


        float dacc;
        //dx update
        {
            float gqr,gql;
            gqr=gqx=gqxpt[pt];
            gql=__shfl_up(gqx,1);
            if (lf)
                gql=gqsave;
            gqsave=__shfl_down(gqx,31);//save for next iter
            dacc = gqr - gql;//dx part
        }
        //dy update and d store
        {
            float a;
            //load
            {
                a=apt[pt];
            }
            float gqu,gqd;
            float d=dpt[pt];
            gqd=gqy=gqypt[pt];
            s[bpt]=gqy;
            __syncthreads();
            if(!btop)
                gqu=s[bpt+bupoff];
            else
                gqu=gqypt[pt + upoff];
            if(y==0)
                gqu=0.0f;
            dacc += gqd-gqu; //dy part
            //d += dacc*.5f;//simplified step
            d = ( d + sigma_d*(dacc + a/theta) ) / (1 + sigma_d/theta);

            dpt[pt] = d;
        }
        __syncthreads();//can't figure out why this is needed, but it is to avoid subtle errors in Qy at the ends of the warp
    }
#endif
}


}}}}
